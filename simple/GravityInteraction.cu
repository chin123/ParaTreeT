#include "hip/hip_runtime.h"
#include "GravityInteraction.h"

#define PARTICLES_PER_THREAD 1
#define BLOCK_SIZE 16

#define gpuSafe(retval) gpuPrintError((retval), __FILE__, __LINE__)
#define gpuCheck() gpuPrintError(hipGetLastError(), __FILE__, __LINE__)

inline void gpuPrintError(hipError_t err, const char *file, int line) {
  if (err != hipSuccess)
    fprintf(stderr,"CUDA Error: %s at %s:%d\n", hipGetErrorString(err), file, line);
}

__global__ void leafKernel(Particle* from_particles, int from_n_particles,
    Particle* on_particles, int on_n_particles, Vector3D<Real>* sum_forces,
    Real gconst, int n_threads) {
  int gx = blockDim.x * blockIdx.x + threadIdx.x;
  int gy = blockDim.y * blockIdx.y + threadIdx.y;
  int gi = gy * (gridDim.x * blockDim.x) + gx; // Linearized global thread index

  if (gi < n_threads) {
    // TODO: Performance is probably abysmal without shared memory
    for (int i = gi * PARTICLES_PER_THREAD; i < (gi + 1) * PARTICLES_PER_THREAD
        && i < on_n_particles; i++) {
      for (int j = 0; j < from_n_particles; j++) {
        if (on_particles[i].key == on_particles[j].key) {
          continue;
        }

        Vector3D<Real>& from_pos = from_particles[j].position;
        Vector3D<Real>& on_pos = on_particles[i].position;

        Real rsq = (from_pos - on_pos).lengthSquared();
        sum_forces[i] += (from_pos - on_pos) * gconst * from_particles[j].mass
          * on_particles[i].mass / (rsq * sqrt(rsq));
      }
    }
  }
}

__global__ void nonLeafKernel(Vector3D<Real> from_centroid, Real from_sum_mass,
    Particle* on_particles, int on_n_particles, Vector3D<Real>* sum_forces,
    Real gconst, int n_threads) {
  int gx = blockDim.x * blockIdx.x + threadIdx.x;
  int gy = blockDim.y * blockIdx.y + threadIdx.y;
  int gi = gy * (gridDim.x * blockDim.x) + gx; // Linearized global thread index

  if (gi < n_threads) {
    // TODO: Performance is probably abysmal without shared memory
    for (int i = gi * PARTICLES_PER_THREAD; i < (gi + 1) * PARTICLES_PER_THREAD
        && i < on_n_particles; i++) {
      Vector3D<Real>& on_pos = on_particles[i].position;

      Real rsq = (from_centroid - on_pos).lengthSquared();
      sum_forces[i] += (from_centroid - on_pos) * gconst * from_sum_mass
        * on_particles[i].mass / (rsq * sqrt(rsq));
    }
  }
}

void invokeKernel(Particle* from_particles, int from_n_particles,
    const Vector3D<Real>& from_centroid, Real from_sum_mass, Particle* on_particles,
    int on_n_particles, Vector3D<Real>* sum_forces, Real gconst, bool is_leaf) {
  Particle* d_from_particles;
  Vector3D<Real>* d_from_centroid;
  Particle* d_on_particles;
  Vector3D<Real>* d_sum_forces;

  // TODO: Move hipMalloc and cudaFrees out if possible (important for multiple iterations)
  gpuSafe(hipMalloc(&d_from_particles, sizeof(Particle) * from_n_particles));
  //gpuSafe(hipMalloc(&d_from_centroid, sizeof(Vector3D<Real>)));
  gpuSafe(hipMalloc(&d_on_particles, sizeof(Particle) * on_n_particles));
  gpuSafe(hipMalloc(&d_sum_forces, sizeof(Vector3D<Real>) * on_n_particles));

  int n_threads = ceil((double)on_n_particles / PARTICLES_PER_THREAD);
  int n_threads_per_dim = ceil(sqrt((double)n_threads));
  int grid_size = ceil((double)n_threads_per_dim / BLOCK_SIZE);

  dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid_dim(grid_size, grid_size);

  if (is_leaf) {
    // TODO: Only transfer necessary portions of particles
    // TODO: Use hipMemcpyAsync instead
    gpuSafe(hipMemcpy(d_from_particles, from_particles, sizeof(Particle)
          * from_n_particles, hipMemcpyHostToDevice));
    //gpuSafe(hipMemcpy(d_from_centroid, &from_centroid, sizeof(Vector3D<Real>),
    //      hipMemcpyHostToDevice));
    gpuSafe(hipMemcpy(d_on_particles, on_particles, sizeof(Particle)
          * on_n_particles, hipMemcpyHostToDevice));
    gpuSafe(hipMemcpy(d_sum_forces, sum_forces, sizeof(Vector3D<Real>)
          * on_n_particles, hipMemcpyHostToDevice));

    // TODO: Use CUDA stream
    leafKernel<<<grid_dim, block_dim>>>(d_from_particles, from_n_particles,
        d_on_particles, on_n_particles, d_sum_forces, gconst, n_threads);
    gpuCheck();

    gpuSafe(hipMemcpy(sum_forces, d_sum_forces, sizeof(Vector3D<Real>)
          * on_n_particles, hipMemcpyDeviceToHost));
  }
  else {
    // TODO: Only transfer necessary portions of particles
    // TODO: Use hipMemcpyAsync instead
    //gpuSafe(hipMemcpy(d_from_centroid, &from_centroid, sizeof(Vector3D<Real>),
    //      hipMemcpyHostToDevice));
    gpuSafe(hipMemcpy(d_on_particles, on_particles, sizeof(Particle)
          * on_n_particles, hipMemcpyHostToDevice));
    gpuSafe(hipMemcpy(d_sum_forces, sum_forces, sizeof(Vector3D<Real>)
          * on_n_particles, hipMemcpyHostToDevice));

    // TODO: Use CUDA stream
    nonLeafKernel<<<grid_dim, block_dim>>>(from_centroid, from_sum_mass,
        d_on_particles, on_n_particles, d_sum_forces, gconst, n_threads);
    gpuCheck();

    gpuSafe(hipMemcpy(sum_forces, d_sum_forces, sizeof(Vector3D<Real>)
          * on_n_particles, hipMemcpyDeviceToHost));
  }

  gpuSafe(hipFree(d_from_particles));
  //gpuSafe(hipFree(d_from_centroid));
  gpuSafe(hipFree(d_on_particles));
  gpuSafe(hipFree(d_sum_forces));
}
